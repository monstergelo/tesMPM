#include "Material.cuh"

#include <hip/hip_runtime.h>

__host__ Material::Material() : mass(1), restDensity(2), stiffness(1), bulkViscosity(1), surfaceTension(0), kElastic(0), maxDeformation(0), meltRate(0), viscosity(.02), damping(.001), friction(0), stickiness(0), smoothing(.02), gravity(.03) {};
